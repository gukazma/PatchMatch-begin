#include "hip/hip_runtime.h"
#include "PatchMatch.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <>

__global__ void squareKernel(float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        output[idx] = input[idx] * input[idx];
    }
}


void PatchMatch::Run()
{
    const int size = 10;
    float* h_input, * h_output;  // Host arrays
    float* d_input, * d_output;  // Device arrays

    // Allocate memory on the host
    h_input = (float*)malloc(size * sizeof(float));
    h_output = (float*)malloc(size * sizeof(float));

    // Initialize input data
    for (int i = 0; i < size; i++) {
        h_input[i] = i;
    }

    // Allocate memory on the device
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    squareKernel << <numBlocks, blockSize >> > (d_input, d_output, size);

    // Copy the result back from device to host
    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; i++) {
        printf("%f ", h_output[i]);
    }

    // Free memory
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
}