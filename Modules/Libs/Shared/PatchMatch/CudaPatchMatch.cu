#include "hip/hip_runtime.h"
#include "CudaPatchMatch.h"
#include <iostream>
#include <unordered_set>
#include <hip/hip_runtime.h>
#include <>
#include <Eigen/Core>
#include <colmap/math/math.h>
#include <colmap/util/misc.h>
#include <colmap/util/hip/hip_runtime.h>
#include <colmap/util/cudacc.h>
#include <colmap/mvs/gpu_mat_ref_image.h>
#include <colmap/mvs/gpu_mat.h>
#include <colmap/mvs/gpu_mat_prng.h>

#define PrintOption(option) LOG(INFO) << #option ": " << option << std::endl
namespace GU
{
    void CudaPatchMatch::Problem::Print() const {
        colmap::PrintHeading2("PatchMatch::Problem");

        PrintOption(ref_image_idx);

        LOG(INFO) << "src_image_idxs: ";
        if (!src_image_idxs.empty()) {
            for (size_t i = 0; i < src_image_idxs.size() - 1; ++i) {
                LOG(INFO) << src_image_idxs[i] << " ";
            }
            LOG(INFO) << src_image_idxs.back();
        }
        else {
        }
    }
    CudaPatchMatch::CudaPatchMatch(Options options_, Problem problem_)
        : m_options(options_), m_problem(problem_)
    {
        colmap::SetBestCudaDevice(std::stoi(options_.gpu_index));
        InitRefImage();
        InitSourceImages();
        InitTransforms();
        InitWorkspaceMemory();
    }

    CudaPatchMatch::~CudaPatchMatch()
    {}
    void CudaPatchMatch::Check() const {
        CHECK(m_options.Check());

        CHECK(!m_options.gpu_index.empty());
        const std::vector<int> gpu_indices = colmap::CSVToVector<int>(m_options.gpu_index);
        CHECK_EQ(gpu_indices.size(), 1);
        CHECK_GE(gpu_indices[0], -1);

        CHECK_NOTNULL(m_problem.images);
        if (m_options.geom_consistency) {
            CHECK_NOTNULL(m_problem.depth_maps);
            CHECK_NOTNULL(m_problem.normal_maps);
            CHECK_EQ(m_problem.depth_maps->size(), m_problem.images->size());
            CHECK_EQ(m_problem.normal_maps->size(), m_problem.images->size());
        }

        CHECK_GT(m_problem.src_image_idxs.size(), 0);

        // Check that there are no duplicate images and that the reference image
        // is not defined as a source image.
        std::set<int> unique_image_idxs(m_problem.src_image_idxs.begin(),
            m_problem.src_image_idxs.end());
        unique_image_idxs.insert(m_problem.ref_image_idx);
        CHECK_EQ(m_problem.src_image_idxs.size() + 1, unique_image_idxs.size());

        // Check that input data is well-formed.
        for (const int image_idx : unique_image_idxs) {
            CHECK_GE(image_idx, 0) << image_idx;
            CHECK_LT(image_idx, m_problem.images->size()) << image_idx;

            const colmap::mvs::Image& image = m_problem.images->at(image_idx);
            CHECK_GT(image.GetBitmap().Width(), 0) << image_idx;
            CHECK_GT(image.GetBitmap().Height(), 0) << image_idx;
            CHECK(image.GetBitmap().IsGrey()) << image_idx;
            CHECK_EQ(image.GetWidth(), image.GetBitmap().Width()) << image_idx;
            CHECK_EQ(image.GetHeight(), image.GetBitmap().Height()) << image_idx;

            // Make sure, the calibration matrix only contains fx, fy, cx, cy.
            CHECK_LT(std::abs(image.GetK()[1] - 0.0f), 1e-6f) << image_idx;
            CHECK_LT(std::abs(image.GetK()[3] - 0.0f), 1e-6f) << image_idx;
            CHECK_LT(std::abs(image.GetK()[6] - 0.0f), 1e-6f) << image_idx;
            CHECK_LT(std::abs(image.GetK()[7] - 0.0f), 1e-6f) << image_idx;
            CHECK_LT(std::abs(image.GetK()[8] - 1.0f), 1e-6f) << image_idx;

            if (m_options.geom_consistency) {
                CHECK_LT(image_idx, m_problem.depth_maps->size()) << image_idx;
                const colmap::mvs::DepthMap& depth_map = m_problem.depth_maps->at(image_idx);
                CHECK_EQ(image.GetWidth(), depth_map.GetWidth()) << image_idx;
                CHECK_EQ(image.GetHeight(), depth_map.GetHeight()) << image_idx;
            }
        }

        if (m_options.geom_consistency) {
            const colmap::mvs::Image& ref_image = m_problem.images->at(m_problem.ref_image_idx);
            const colmap::mvs::NormalMap& ref_normal_map =
                m_problem.normal_maps->at(m_problem.ref_image_idx);
            CHECK_EQ(ref_image.GetWidth(), ref_normal_map.GetWidth());
            CHECK_EQ(ref_image.GetHeight(), ref_normal_map.GetHeight());
        }
    }
    void CudaPatchMatch::ComputeCudaConfig()
    {
    }
    void CudaPatchMatch::BindRefImageTexture()
    {
    }
    void CudaPatchMatch::InitRefImage()
    {
        const colmap::mvs::Image& ref_image = m_problem.images->at(m_problem.ref_image_idx);

        m_refWidth = ref_image.GetWidth();
        m_refHeight= ref_image.GetHeight();

        // Upload to device and filter.
        m_refImage.reset(new colmap::mvs::GpuMatRefImage(m_refWidth, m_refHeight));
        const std::vector<uint8_t> ref_image_array =
            ref_image.GetBitmap().ConvertToRowMajorArray();
        // ͼ���˫���˲�
        m_refImage->Filter(ref_image_array.data(),
            m_options.window_radius,
            m_options.window_step,
            m_options.sigma_spatial,
            m_options.sigma_color);

        BindRefImageTexture();
    }
    void CudaPatchMatch::InitSourceImages()
    {
        // Determine maximum image size.
        size_t max_width = 0;
        size_t max_height = 0;
        for (const auto image_idx : m_problem.src_image_idxs) {
            const colmap::mvs::Image& image = m_problem.images->at(image_idx);
            if (image.GetWidth() > max_width) {
                max_width = image.GetWidth();
            }
            if (image.GetHeight() > max_height) {
                max_height = image.GetHeight();
            }
        }

        // Upload source images to device.
        {
            // Copy source images to contiguous memory block.
            const uint8_t kDefaultValue = 0;
            std::vector<uint8_t> src_images_host_data(
                static_cast<size_t>(max_width * max_height *
                    m_problem.src_image_idxs.size()),
                kDefaultValue);
            for (size_t i = 0; i < m_problem.src_image_idxs.size(); ++i) {
                const colmap::mvs::Image& image = m_problem.images->at(m_problem.src_image_idxs[i]);
                const colmap::Bitmap& bitmap = image.GetBitmap();
                uint8_t* dest = src_images_host_data.data() + max_width * max_height * i;
                for (size_t r = 0; r < image.GetHeight(); ++r) {
                    memcpy(dest, bitmap.GetScanline(r), image.GetWidth() * sizeof(uint8_t));
                    dest += max_width;
                }
            }

            // Create source images texture.
            hipTextureDesc texture_desc;
            memset(&texture_desc, 0, sizeof(texture_desc));
            texture_desc.addressMode[0] = hipAddressModeBorder;
            texture_desc.addressMode[1] = hipAddressModeBorder;
            texture_desc.addressMode[2] = hipAddressModeBorder;
            texture_desc.filterMode = hipFilterModeLinear;
            texture_desc.readMode = hipReadModeNormalizedFloat;
            texture_desc.normalizedCoords = false;
            m_srcImagesTexture = colmap::mvs::CudaArrayLayeredTexture<uint8_t>::FromHostArray(
                texture_desc,
                max_width,
                max_height,
                m_problem.src_image_idxs.size(),
                src_images_host_data.data());
        }

        // Upload source depth maps to device.
        if (m_options.geom_consistency) {
            const float kDefaultValue = 0.0f;
            std::vector<float> src_depth_maps_host_data(
                static_cast<size_t>(max_width * max_height *
                    m_problem.src_image_idxs.size()),
                kDefaultValue);
            for (size_t i = 0; i < m_problem.src_image_idxs.size(); ++i) {
                const colmap::mvs::DepthMap& depth_map =
                    m_problem.depth_maps->at(m_problem.src_image_idxs[i]);
                float* dest =
                    src_depth_maps_host_data.data() + max_width * max_height * i;
                for (size_t r = 0; r < depth_map.GetHeight(); ++r) {
                    memcpy(dest,
                        depth_map.GetPtr() + r * depth_map.GetWidth(),
                        depth_map.GetWidth() * sizeof(float));
                    dest += max_width;
                }
            }

            // Create source depth maps texture.
            hipTextureDesc texture_desc;
            memset(&texture_desc, 0, sizeof(texture_desc));
            texture_desc.addressMode[0] = hipAddressModeBorder;
            texture_desc.addressMode[1] = hipAddressModeBorder;
            texture_desc.addressMode[2] = hipAddressModeBorder;
            texture_desc.filterMode = hipFilterModePoint;
            texture_desc.readMode = hipReadModeElementType;
            texture_desc.normalizedCoords = false;
            m_srcDepthMapsTexture = colmap::mvs::CudaArrayLayeredTexture<float>::FromHostArray(
                texture_desc,
                max_width,
                max_height,
                m_problem.src_image_idxs.size(),
                src_depth_maps_host_data.data());
        }
    }
    void CudaPatchMatch::InitTransforms()
    {
    }
    void CudaPatchMatch::InitWorkspaceMemory()
    {
    }
    void CudaPatchMatch::Rotate()
    {
    }
    void CudaPatchMatch::Run()
    {
        colmap::PrintHeading2("PatchMatch::Run");
        Check();
    }
    void GU::CudaPatchMatch::Options::Print() const
    {
        colmap::PrintHeading2("PatchMatchOptions");
        PrintOption(max_image_size);
        PrintOption(gpu_index);
        PrintOption(depth_min);
        PrintOption(depth_max);
        PrintOption(window_radius);
        PrintOption(window_step);
        PrintOption(sigma_spatial);
        PrintOption(sigma_color);
        PrintOption(num_samples);
        PrintOption(ncc_sigma);
        PrintOption(min_triangulation_angle);
        PrintOption(incident_angle_sigma);
        PrintOption(num_iterations);
        PrintOption(geom_consistency);
        PrintOption(geom_consistency_regularizer);
        PrintOption(geom_consistency_max_cost);
        PrintOption(filter);
        PrintOption(filter_min_ncc);
        PrintOption(filter_min_triangulation_angle);
        PrintOption(filter_min_num_consistent);
        PrintOption(filter_geom_consistency_max_cost);
        PrintOption(write_consistency_graph);
        PrintOption(allow_missing_files);
    }

    bool CudaPatchMatch::Options::Check() const
    {
        using namespace colmap;
        if (depth_min != -1.0f || depth_max != -1.0f) {
            CHECK_OPTION_LE(depth_min, depth_max);
            CHECK_OPTION_GE(depth_min, 0.0f);
        }
        CHECK_OPTION_LE(window_radius,
            static_cast<int>(kMaxPatchMatchWindowRadius));
        CHECK_OPTION_GT(sigma_color, 0.0f);
        CHECK_OPTION_GT(window_radius, 0);
        CHECK_OPTION_GT(window_step, 0);
        CHECK_OPTION_LE(window_step, 2);
        CHECK_OPTION_GT(num_samples, 0);
        CHECK_OPTION_GT(ncc_sigma, 0.0f);
        CHECK_OPTION_GE(min_triangulation_angle, 0.0f);
        CHECK_OPTION_LT(min_triangulation_angle, 180.0f);
        CHECK_OPTION_GT(incident_angle_sigma, 0.0f);
        CHECK_OPTION_GT(num_iterations, 0);
        CHECK_OPTION_GE(geom_consistency_regularizer, 0.0f);
        CHECK_OPTION_GE(geom_consistency_max_cost, 0.0f);
        CHECK_OPTION_GE(filter_min_ncc, -1.0f);
        CHECK_OPTION_LE(filter_min_ncc, 1.0f);
        CHECK_OPTION_GE(filter_min_triangulation_angle, 0.0f);
        CHECK_OPTION_LE(filter_min_triangulation_angle, 180.0f);
        CHECK_OPTION_GE(filter_min_num_consistent, 0);
        CHECK_OPTION_GE(filter_geom_consistency_max_cost, 0.0f);
        CHECK_OPTION_GT(cache_size, 0);
        return true;
    }

}