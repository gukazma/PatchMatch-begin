#include "CudaPatchMatch.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <>
#include <Eigen/Core>
#include <colmap/math/math.h>
#include <unordered_set>
#include <colmap/util/misc.h>

#define PrintOption(option) LOG(INFO) << #option ": " << option << std::endl
namespace GU
{
    void CudaPatchMatch::Problem::Print() const {
        colmap::PrintHeading2("PatchMatch::Problem");

        PrintOption(ref_image_idx);

        LOG(INFO) << "src_image_idxs: ";
        if (!src_image_idxs.empty()) {
            for (size_t i = 0; i < src_image_idxs.size() - 1; ++i) {
                LOG(INFO) << src_image_idxs[i] << " ";
            }
            LOG(INFO) << src_image_idxs.back();
        }
        else {
        }
    }
    CudaPatchMatch::CudaPatchMatch(Options options_, Problem problem_)
        : m_options(options_), m_problem(problem_)
    {}

    CudaPatchMatch::~CudaPatchMatch()
    {}

    void CudaPatchMatch::Run()
    {}
    void GU::CudaPatchMatch::Options::Print() const
    {
    }

}